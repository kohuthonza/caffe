
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <cmath>
#include <vector>

#include "caffe/layers/cudnn_binary_conv_layer.hpp"

namespace caffe {

__global__ void sync_binary_conv_groups() { }


template <typename Dtype>
void CuDNNBinaryConvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = this->blobs_[0]->cpu_data();
  Dtype* mutable_binary_weight = this->binary_weight_->mutable_cpu_data();
  this->compute_binary_weight(weight, mutable_binary_weight, this->compute_kernel_alfa(weight));
  const Dtype* binary_weight = this->binary_weight_->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();

    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      // Filters.
      CUDNN_CHECK(cudnnConvolutionForward(handle_[g],
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data + bottom_offset_ * g,
            filter_desc_, binary_weight + this->weight_offset_ * g,
            conv_descs_[i],
            fwd_algo_[i], workspace[g], workspace_fwd_sizes_[i],
            cudnn::dataType<Dtype>::zero,
            top_descs_[i], top_data + top_offset_ * g));

      // Bias.
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
        CUDNN_CHECK(cudnnAddTensor(handle_[g],
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_data + top_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_binary_conv_groups<<<1, 1>>>();
  }
}

template <typename Dtype>
void CuDNNBinaryConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  Dtype* mutable_binary_weight = NULL;
  const Dtype* binary_weight = NULL;
  Dtype* binary_weight_diff = NULL;
  vector<Dtype> kernel_alfa;
  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->cpu_data();
    if (this->gradient_update_) {
      weight_diff = this->blobs_[0]->mutable_cpu_diff();
    }
    else {
      weight_diff = this->blobs_[0]->mutable_gpu_diff();
    }
    mutable_binary_weight = this->binary_weight_->mutable_cpu_data();
    kernel_alfa = this->compute_kernel_alfa(weight);
    this->compute_binary_weight(weight, mutable_binary_weight, kernel_alfa);
    binary_weight = this->binary_weight_->gpu_data();
    if (this->gradient_update_) {
      binary_weight_diff = this->binary_weight_->mutable_cpu_diff();
      for (int i = 0; i < this->binary_weight_->count(); ++i) {
        binary_weight_diff[i] = 0.;
      }
      binary_weight_diff = this->binary_weight_->mutable_gpu_diff();
    }
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
  }
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[0*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              top_descs_[i],  top_diff + top_offset_ * g,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_diff + bias_offset_ * g));
      }

      // Gradient w.r.t. weights.

      if (this->param_propagate_down_[0]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        if (this->gradient_update_) {
          CUDNN_CHECK(cudnnConvolutionBackwardFilter(
                handle_[1*this->group_ + g],
                cudnn::dataType<Dtype>::one,
                bottom_descs_[i], bottom_data + bottom_offset_ * g,
                top_descs_[i],    top_diff + top_offset_ * g,
                conv_descs_[i],
                bwd_filter_algo_[i], workspace[1*this->group_ + g],
                workspace_bwd_filter_sizes_[i],
                cudnn::dataType<Dtype>::one,
                filter_desc_, binary_weight_diff + this->weight_offset_ * g));
        }
        else {
          CUDNN_CHECK(cudnnConvolutionBackwardFilter(
                handle_[1*this->group_ + g],
                cudnn::dataType<Dtype>::one,
                bottom_descs_[i], bottom_data + bottom_offset_ * g,
                top_descs_[i],    top_diff + top_offset_ * g,
                conv_descs_[i],
                bwd_filter_algo_[i], workspace[1*this->group_ + g],
                workspace_bwd_filter_sizes_[i],
                cudnn::dataType<Dtype>::one,
                filter_desc_, weight_diff + this->weight_offset_ * g));
        }
      }


      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        if (weight == NULL) {
          weight = this->blobs_[0]->cpu_data();
          mutable_binary_weight = this->binary_weight_->mutable_cpu_data();
          kernel_alfa = this->compute_kernel_alfa(weight);
          this->compute_binary_weight(weight, mutable_binary_weight, kernel_alfa);
          binary_weight = this->binary_weight_->gpu_data();
        }
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        CUDNN_CHECK(cudnnConvolutionBackwardData(
              handle_[2*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              filter_desc_, binary_weight + this->weight_offset_ * g,
              top_descs_[i], top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_data_algo_[i], workspace[2*this->group_ + g],
              workspace_bwd_data_sizes_[i],
              cudnn::dataType<Dtype>::zero,
              bottom_descs_[i], bottom_diff + bottom_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_binary_conv_groups<<<1, 1>>>();
  }
  if (this->gradient_update_) {
    binary_weight_diff = this->binary_weight_->mutable_cpu_diff();
    this->compute_binary_weight_diff(weight, weight_diff, binary_weight_diff, kernel_alfa);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNBinaryConvolutionLayer);

}  // namespace caffe
#endif
