
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <cmath>
#include <vector>

#include "caffe/layers/cudnn_binary_conv_layer.hpp"

#include <iostream>
using namespace std;

namespace caffe {

__global__ void sync_binary_conv_groups() { }

template <typename Dtype>
__global__ void copy_abs_value(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = abs(in[index]);
  }
}

template <typename Dtype>
__global__ void multiply_alfa_with_binary_weight(const int n, const Dtype* weight,
               Dtype* binary_weight, const Dtype* alfa_kernel, const int kernel_size) {
  CUDA_KERNEL_LOOP(index, n) {
    binary_weight[index] = copysign(1.0, weight[index]) * alfa_kernel[index / kernel_size];
  }
}

template <typename Dtype>
__global__ void set_values_to_zero(const int n, Dtype* in) {
  CUDA_KERNEL_LOOP(index, n) {
    in[index] = 0.;
  }
}

template <typename Dtype>
__global__ void gradient_update(const int n, const Dtype* weight,
              Dtype* weight_diff, Dtype* binary_weight_diff,
              const Dtype* alfa_kernel, const int kernel_size) {
  CUDA_KERNEL_LOOP(index, n) {
    if (weight[index] < 1. && weight[index] > -1.) {
      weight_diff[index] += binary_weight_diff[index] *
                            (alfa_kernel[index / kernel_size] + 1./kernel_size);
    }
    else {
      weight_diff[index] += binary_weight_diff[index] * 1./kernel_size;
    }
  }
}

template <typename Dtype>
__global__ void gradient_scale(const int n, Dtype* weight_diff,
              const int kernel_size) {
  CUDA_KERNEL_LOOP(index, n) {
    weight_diff[index] *= kernel_size;
  }
}


template <typename Dtype>
void CuDNNBinaryConvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* abs_weight = this->abs_weight_->mutable_gpu_data();
  copy_abs_value<<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
                   CAFFE_CUDA_NUM_THREADS>>>
                   (this->blobs_[0]->count(), weight, abs_weight);
  const Dtype* alfa_kernel_multiplier = this->alfa_kernel_multiplier_->gpu_data();
  Dtype* alfa_kernel = this->alfa_kernel_->mutable_gpu_data();
  CUDNN_CHECK(cudnnConvolutionForward(*alfa_handle_,
        cudnn::dataType<Dtype>::one,
        alfa_bottom_desc_, abs_weight,
        alfa_filter_desc_, alfa_kernel_multiplier,
        alfa_conv_desc_,
        *alfa_fwd_algo_, NULL, 0,
        cudnn::dataType<Dtype>::zero,
        alfa_top_desc_, alfa_kernel));
  Dtype* binary_weight = this->binary_weight_->mutable_gpu_data();
  multiply_alfa_with_binary_weight<<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
                          CAFFE_CUDA_NUM_THREADS>>>
                          (this->blobs_[0]->count(), weight, binary_weight,
                           alfa_kernel, this->kernel_size_);
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();

    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      // Filters.
      CUDNN_CHECK(cudnnConvolutionForward(handle_[g],
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data + bottom_offset_ * g,
            filter_desc_, binary_weight + this->weight_offset_ * g,
            conv_descs_[i],
            fwd_algo_[i], workspace[g], workspace_fwd_sizes_[i],
            cudnn::dataType<Dtype>::zero,
            top_descs_[i], top_data + top_offset_ * g));

      // Bias.
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
        CUDNN_CHECK(cudnnAddTensor(handle_[g],
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_data + top_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_binary_conv_groups<<<1, 1>>>();
  }
}

template <typename Dtype>
void CuDNNBinaryConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = NULL;
  Dtype* abs_weight = NULL;
  Dtype* weight_diff = NULL;
  Dtype* binary_weight = NULL;
  Dtype* binary_weight_diff = NULL;
  Dtype* alfa_kernel = NULL;
  const Dtype* alfa_kernel_multiplier = NULL;
  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    abs_weight = this->abs_weight_->mutable_gpu_data();
    copy_abs_value<<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
                     CAFFE_CUDA_NUM_THREADS>>>
                     (this->blobs_[0]->count(), weight, abs_weight);
    alfa_kernel_multiplier = this->alfa_kernel_multiplier_->gpu_data();
    alfa_kernel = this->alfa_kernel_->mutable_gpu_data();
    CUDNN_CHECK(cudnnConvolutionForward(*alfa_handle_,
          cudnn::dataType<Dtype>::one,
          alfa_bottom_desc_, abs_weight,
          alfa_filter_desc_, alfa_kernel_multiplier,
          alfa_conv_desc_,
          *alfa_fwd_algo_, NULL, 0,
          cudnn::dataType<Dtype>::zero,
          alfa_top_desc_, alfa_kernel));
    binary_weight = this->binary_weight_->mutable_gpu_data();
    multiply_alfa_with_binary_weight<<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
                            CAFFE_CUDA_NUM_THREADS>>>
                            (this->blobs_[0]->count(), weight, binary_weight,
                             alfa_kernel, this->kernel_size_);
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
    if (this->gradient_update_) {
      binary_weight_diff = this->binary_weight_->mutable_gpu_diff();
      set_values_to_zero<<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
                           CAFFE_CUDA_NUM_THREADS>>>
                           (this->binary_weight_->count(), binary_weight_diff);
    }
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
  }
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[0*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              top_descs_[i],  top_diff + top_offset_ * g,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_diff + bias_offset_ * g));
      }

      // Gradient w.r.t. weights.

      if (this->param_propagate_down_[0]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        if (this->gradient_update_) {
          CUDNN_CHECK(cudnnConvolutionBackwardFilter(
                handle_[1*this->group_ + g],
                cudnn::dataType<Dtype>::one,
                bottom_descs_[i], bottom_data + bottom_offset_ * g,
                top_descs_[i],    top_diff + top_offset_ * g,
                conv_descs_[i],
                bwd_filter_algo_[i], workspace[1*this->group_ + g],
                workspace_bwd_filter_sizes_[i],
                cudnn::dataType<Dtype>::one,
                filter_desc_, binary_weight_diff + this->weight_offset_ * g));
        }
        else {
          CUDNN_CHECK(cudnnConvolutionBackwardFilter(
                handle_[1*this->group_ + g],
                cudnn::dataType<Dtype>::one,
                bottom_descs_[i], bottom_data + bottom_offset_ * g,
                top_descs_[i],    top_diff + top_offset_ * g,
                conv_descs_[i],
                bwd_filter_algo_[i], workspace[1*this->group_ + g],
                workspace_bwd_filter_sizes_[i],
                cudnn::dataType<Dtype>::one,
                filter_desc_, weight_diff + this->weight_offset_ * g));
        }
      }


      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        if (weight == NULL) {
          const Dtype* weight = this->blobs_[0]->gpu_data();
          Dtype* abs_weight = this->abs_weight_->mutable_gpu_data();
          copy_abs_value<<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
                           CAFFE_CUDA_NUM_THREADS>>>
                           (this->blobs_[0]->count(), weight, abs_weight);
          const Dtype* alfa_kernel_multiplier = this->alfa_kernel_multiplier_->gpu_data();
          Dtype* alfa_kernel = this->alfa_kernel_->mutable_gpu_data();
          CUDNN_CHECK(cudnnConvolutionForward(*alfa_handle_,
                cudnn::dataType<Dtype>::one,
                alfa_bottom_desc_, abs_weight,
                alfa_filter_desc_, alfa_kernel_multiplier,
                alfa_conv_desc_,
                *alfa_fwd_algo_, NULL, 0,
                cudnn::dataType<Dtype>::zero,
                alfa_top_desc_, alfa_kernel));
          Dtype* binary_weight = this->binary_weight_->mutable_gpu_data();
          multiply_alfa_with_binary_weight<<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
                                  CAFFE_CUDA_NUM_THREADS>>>
                                  (this->blobs_[0]->count(), weight, binary_weight,
                                   alfa_kernel, this->kernel_size_);
        }
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        CUDNN_CHECK(cudnnConvolutionBackwardData(
              handle_[2*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              filter_desc_, binary_weight + this->weight_offset_ * g,
              top_descs_[i], top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_data_algo_[i], workspace[2*this->group_ + g],
              workspace_bwd_data_sizes_[i],
              cudnn::dataType<Dtype>::zero,
              bottom_descs_[i], bottom_diff + bottom_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_binary_conv_groups<<<1, 1>>>();
  }
  if (this->gradient_update_) {
    gradient_update<<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
                      CAFFE_CUDA_NUM_THREADS>>>
                      (this->blobs_[0]->count(), weight, weight_diff,
                       binary_weight_diff, alfa_kernel, this->kernel_size_);
    if (this->gradient_scale_) {
      gradient_scale<<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
                       CAFFE_CUDA_NUM_THREADS>>>
                       (this->blobs_[0]->count(), weight_diff, this->kernel_size_);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNBinaryConvolutionLayer);

}  // namespace caffe
#endif
